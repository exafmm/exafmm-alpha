#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#include <cutil.h>

#include <gpuvdwforce_kernel.cu>
////////////////////////////////////////////////////////////////////////////////
extern "C"
void gpuvdwforce_ (double*, int, int*, int, double*, double*, int, double, int, int, double*,
		   double, double);
void gpuvdwforce__ (double*, int, int*, int, double*, double*, int, double, int, int, double*,
		    double *, double *);
//void printDiff(float*, float*, int, int);

extern "C"
//void computeGold2( float*, const double*, const double*, unsigned int, double);
//void computeGold2_d(double*, const double*, const double*, unsigned int, double);
//void computeGold2_d2(double*, const double*, const double*, unsigned int, double);
//void computeGold2_f2(float*, const float*, const float*, unsigned int, float);
////////////////////////////////////////////////////////////////////////////////
//! One dimensional dimensionless Lennard-Jones Simulation
////////////////////////////////////////////////////////////////////////////////
/*
extern "C"
void get_cputime(double *laptime, double *sprittime)
{
  struct timeval tv;
  struct timezone tz;
  double sec,microsec;

  gettimeofday(&tv, &tz);
  sec=tv.tv_sec;
  microsec=tv.tv_usec;

  *sprittime = sec + microsec * 1e-6 - *laptime;
  *laptime = sec + microsec * 1e-6;
}
*/

extern "C"
void
gpuvdwforce_ (double* x, int n, int* atype, int nat_org, double* gscale, double* rscale, int tblno, double xmax, int periodicflag, int natchangeflag, double* force,
	      double r2mind, double r2maxd)
{
#ifndef CUDA_SDK_2
   CUT_DEVICE_INIT();
#endif
//   CUT_CHECK_DEVICE();

   unsigned int size_A = ((n+THD-1)/THD*THD) * 3;
   unsigned int mem_size_A = sizeof(float) * size_A;
   float* x_float = (float*) malloc(mem_size_A);
   float* f_float = (float*) malloc(mem_size_A);
   unsigned int size_B = THD;
   unsigned int mem_size_B = sizeof(float) * size_B;
   float* gr_float = (float*) malloc(mem_size_B);

   //double stime,ltime;
   float r2min=(float)r2mind;
   float r2max=(float)r2maxd;
   int nat=nat_org+1;

   //   printf("n=%d n*3=%d size_A=%d\n",n,n*3,size_A);
   for (int i = 0; i < size_A; i++){
     if(i<n*3) x_float[i] = (float)x[i];
     else      x_float[i]=0.0;
   }
   for(int i=0;i<nat;i++){
     for(int j=0;j<nat;j++){
       if(i<nat_org && j<nat_org){
	 gr_float[(i*nat+j)*2]   = (float)gscale[i*nat_org+j];
	 gr_float[(i*nat+j)*2+1] = (float)rscale[i*nat_org+j];
       }
       else{
	 gr_float[(i*nat+j)*2]   = 0.0f;
	 gr_float[(i*nat+j)*2+1] = 1.0f;
       }
     }
   }
   
   float* d_B;
   CUDA_SAFE_CALL(hipMalloc((void**) &d_B, mem_size_B));
   CUDA_SAFE_CALL(hipMemcpy(d_B, gr_float, mem_size_B,hipMemcpyHostToDevice) );
   float* d_A;
   CUDA_SAFE_CALL(hipMalloc((void**) &d_A, mem_size_A));
   CUDA_SAFE_CALL(hipMemcpy(d_A, x_float, mem_size_A,hipMemcpyHostToDevice) );
   unsigned int size_D = (n+THD-1)/THD*THD;
   unsigned int mem_size_D = sizeof(int) * size_D;
   int* d_D;
   int *atype2 = (int *)malloc(mem_size_D);

   for(int i=0;i<size_D;i++){
     if(i<n) atype2[i]=atype[i];
     else    atype2[i]=nat_org;
   }
   CUDA_SAFE_CALL(hipMalloc((void**) &d_D, mem_size_D));
   CUDA_SAFE_CALL(hipMemcpy(d_D, atype2, mem_size_D,hipMemcpyHostToDevice) );

   unsigned int size_C = size_A;
   unsigned int mem_size_C = sizeof(float) * size_C;
   float* d_C;
   CUDA_SAFE_CALL(hipMalloc((void**) &d_C, mem_size_C));

   //get_cputime(&ltime,&stime);

   float xmax_float = (float)xmax;

   dim3 threads(THD);
   dim3 grid((n+THD-1)/THD);
   gpuvdwforce_kernel<<< grid, threads >>>(d_C, d_A, d_B, d_D, n, xmax_float, nat,
					   r2min,r2max);

   CUT_CHECK_ERROR("Kernel execution failed");

   CUDA_SAFE_CALL(hipMemcpy(f_float, d_C, mem_size_C,hipMemcpyDeviceToHost) );
   //get_cputime(&ltime,&stime);
   //printf("GPU  Processing time: %10.3f (sec)\n", stime);
   /*
   float* reference = (float*) malloc(mem_size_A);
   for (int i = 0; i < n; ++i){
     reference[i*3] = 0.e0;
     reference[i*3+1] = 0.e0;
     reference[i*3+2] = 0.e0;
   }
   */
   //get_cputime(&ltime,&stime);
   //computeGold2_f2(reference, x_float, x_float, n, xmax_float);
   
   //computeGold(reference, x, x, n, xmax);
   /*
   unsigned int size_E = n * 3;
   unsigned int mem_size_E = size_E * sizeof(double);
   double* force_double = (double*) malloc(mem_size_E);

   for (int i = 0; i < n; ++i){
     force_double[i*3] = 0.e0;
     force_double[i*3+1] = 0.e0;
     force_double[i*3+2] = 0.e0;
     }*/
   //computeGold2_d2(force_double, x, x, n, xmax);
   
   //get_cputime(&ltime,&stime);
   //printf("HOST Processing time: %10.3f (sec)\n", stime);

   //double sum_gpu = 0.e0;
   //double sum_host = 0.e0;
   
   for (int i = 0; i < n; ++i){
     //sum_gpu += (double)f_float[i*3+1];
     //sum_host += force_double[i*3+1];
     //sum_host += (double)reference[i*3+1];
     //printf("%16.6f %16.6f %d \n",h_C[i*3],reference[i*3],i);

     //printf("%16.6f %16.6f %d \n",f_float[i*3],f_float[i*3+1],i);
     
     force[i*3]   += (double)f_float[i*3];
     force[i*3+1] += (double)f_float[i*3+1];
     force[i*3+2] += (double)f_float[i*3+2];
     
     /*
     force[i*3]   += force_double[i*3];
     force[i*3+1] += force_double[i*3+1];
     force[i*3+2] += force_double[i*3+2];
     */
   }
   
   //printf("HOST: %20.8f  \n",force_double[5]);
   //printf("HOST: %20.8f  \n",reference[5]);
   //printf("GPU : %20.8f  \n",f_float[5]);
   

   free(x_float);
   free(f_float);
   free(gr_float);
   //free(force_double);
   //free(reference);
   CUDA_SAFE_CALL(hipFree(d_A));
   CUDA_SAFE_CALL(hipFree(d_B));
   CUDA_SAFE_CALL(hipFree(d_C));
   CUDA_SAFE_CALL(hipFree(d_D));
}

/*
extern "C"
void
gpuvdwforce_ (double* x, int *n, int* atype, int *nat, double* gscale, double* rscale, int *tblno, double *xmax, int *periodicflag, int *natchangeflag, double* force)
{
  gpuvdwforce_ (x,*n,atype,*nat,gscale,rscale,*tblno,*xmax,*periodicflag,*natchangeflag,force);
}
*/

extern "C"
void
gpuvdwforce__ (double* x, int *n, int* atype, int *nat, double* gscale, double* rscale, int *tblno, double *xmax, int *periodicflag, int *natchangeflag, double* force,
	       double *r2mind, double *r2maxd)
{
  gpuvdwforce_ (x,*n,atype,*nat,gscale,rscale,*tblno,*xmax,*periodicflag,*natchangeflag,force,
		*r2mind,*r2maxd);
}

/*
extern "C"
void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) {
    for (i=0; i<width; i++) {
      k = j*width+i;
      if (data1[k] != data2[k]) {
	printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f\n", i,j, data1[k], data2[k]);
	error_count++;
      }
    }
  }
  printf("\nTotal Errors = %d\n", error_count);
}
*/

void
computeGold2(float* C, const double* A, const double* B, unsigned int num_a, double xmax)
{

  double dn2, adn2, tmp2;
  double dx, dy, dz;
  double f, fx, fy, fz;
  double l2 = 0.5e0 * xmax;
  double exclude_radius2 = 0.1;

  for (unsigned int i = 0; i < num_a; ++i){
    fx = 0.e0;
    fy = 0.e0;
    fz = 0.e0;
    for (unsigned int j = 0; j < num_a; ++j) {
      dx = (A[i*3]   - A[j*3]  );
      dy = (A[i*3+1] - A[j*3+1]);
      dz = (A[i*3+2] - A[j*3+2]);

      if (!(dx < l2 && dx > -l2))
	if (dx > l2){
	  dx = dx - xmax;
	}else{
	  dx = dx + xmax;
	}
      if (!(dy < l2 && dy > -l2))
	if (dy > l2){
	  dy = dy - xmax;
	}else{
	  dy = dy + xmax;
	}
      if (!(dz < l2 && dz > -l2))
	if (dz > l2){
	  dz = dz - xmax;
	}else{
	  dz = dz + xmax;
	}

      dn2 = (dx * dx + dy * dy + dz * dz) * 1;
      if (dn2 > exclude_radius2){
	adn2 = 1.e0 / dn2;
	tmp2 = adn2 * adn2 * adn2;
	f = 24.e0 * adn2 * tmp2 * (2.e0 * tmp2 - 1.e0);
	fx += f * dx;
	fy += f * dy;
	fz += f * dz;
	
      }
    }
    C[i*3]   = (float)fx;
    C[i*3+1] = (float)fy;
    C[i*3+2] = (float)fz;
  }
}

void
computeGold2_d(double* C, const double* A, const double* B, unsigned int num_a, double xmax)
{

  double dn2, tmp2;
  double dx, dy, dz;
  double f, fx, fy, fz;
  double l2 = 0.5e0 * xmax;
  //double exclude_radius2 = 0.01e0;
  double cutoff_radius2 = 9.e0;

  for (unsigned int i = 0; i < num_a; ++i){
    fx = 0.e0;
    fy = 0.e0;
    fz = 0.e0;
    for (unsigned int j = 0; j < num_a; ++j) {

      dx = (A[i*3]   - A[j*3]  );
      dy = (A[i*3+1] - A[j*3+1]);
      dz = (A[i*3+2] - A[j*3+2]);

      if (!(dx < l2 && dx > -l2))
	if (dx > l2){
	  dx = dx - xmax;
	}else{
	  dx = dx + xmax;
	}
      if (!(dy < l2 && dy > -l2))
	if (dy > l2){
	  dy = dy - xmax;
	}else{
	  dy = dy + xmax;
	}
      if (!(dz < l2 && dz > -l2))
	if (dz > l2){
	  dz = dz - xmax;
	}else{
	  dz = dz + xmax;
	}

      dn2 = dx * dx + dy * dy + dz * dz;
      if ((i != j) && dn2 < cutoff_radius2){
	tmp2 = 2.e0 / dn2 / dn2 / dn2 / dn2 / dn2 / dn2 / dn2
	  - 1.e0 / dn2 / dn2 / dn2 / dn2;
	f = tmp2 * 24.e0;
	fx += f * dx;
	fy += f * dy;
	fz += f * dz;

	
      }
      /*
      dn2 = (dx * dx + dy * dy + dz * dz) * 1;
      if (dn2 > exclude_radius2 && dn2 < cutoff_radius2){
	adn2 = 1.e0 / dn2;
	tmp2 = adn2 * adn2 * adn2;
	f = 24.e0 * adn2 * tmp2 * (2.e0 * tmp2 - 1.e0);
	fx += f * dx;
	fy += f * dy;
	fz += f * dz;
	
	}*/
    }
    C[i*3]   = fx;
    C[i*3+1] = fy;
    C[i*3+2] = fz;
	if (fz > 100.e0)
	  printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
  }
}
void
computeGold2_d2(double* C, const double* A, const double* B, unsigned int num_a, double xmax)
{

  double dn2, tmp2;
  double dx, dy, dz;
  double f;
  double l2 = 0.5e0 * xmax;
  //double exclude_radius2 = 0.01e0;
  double cutoff_radius2 = 9.e0;

  for (unsigned int i = 0; i < num_a-1; ++i){
    for (unsigned int j = i+1; j < num_a; ++j) {

      dx = (A[i*3]   - A[j*3]  );
      dy = (A[i*3+1] - A[j*3+1]);
      dz = (A[i*3+2] - A[j*3+2]);

      if (!(dx < l2 && dx > -l2))
	if (dx > l2){
	  dx = dx - xmax;
	}else{
	  dx = dx + xmax;
	}
      if (!(dy < l2 && dy > -l2))
	if (dy > l2){
	  dy = dy - xmax;
	}else{
	  dy = dy + xmax;
	}
      if (!(dz < l2 && dz > -l2))
	if (dz > l2){
	  dz = dz - xmax;
	}else{
	  dz = dz + xmax;
	}

      dn2 = dx * dx + dy * dy + dz * dz;
      if (dn2 < cutoff_radius2){
	tmp2 = 2.e0 / dn2 / dn2 / dn2 / dn2 / dn2 / dn2 / dn2
	  - 1.e0 / dn2 / dn2 / dn2 / dn2;
	f = tmp2 * 24.e0;
	C[i*3]   += f * dx;
	C[i*3+1] += f * dy;
	C[i*3+2] += f * dz;
	C[j*3]   -= f * dx;
	C[j*3+1] -= f * dy;
	C[j*3+2] -= f * dz;
	
      }
      /*
      dn2 = (dx * dx + dy * dy + dz * dz) * 1;
      if (dn2 > exclude_radius2 && dn2 < cutoff_radius2){
	adn2 = 1.e0 / dn2;
	tmp2 = adn2 * adn2 * adn2;
	f = 24.e0 * adn2 * tmp2 * (2.e0 * tmp2 - 1.e0);
	fx += f * dx;
	fy += f * dy;
	fz += f * dz;
	
	}*/
    }
  }
}
void
computeGold2_f2(float* C, const float* A, const float* B, unsigned int num_a, float xmax)
{

  float dn2, tmp2;
  float dx, dy, dz;
  float f;
  float l2 = 0.5e0 * xmax;
  //double exclude_radius2 = 0.01e0;
  float cutoff_radius2 = 9.e0;

  for (unsigned int i = 0; i < num_a-1; ++i){
    for (unsigned int j = i+1; j < num_a; ++j) {

      dx = (A[i*3]   - A[j*3]  );
      dy = (A[i*3+1] - A[j*3+1]);
      dz = (A[i*3+2] - A[j*3+2]);

      if (!(dx < l2 && dx > -l2))
	if (dx > l2){
	  dx = dx - xmax;
	}else{
	  dx = dx + xmax;
	}
      if (!(dy < l2 && dy > -l2))
	if (dy > l2){
	  dy = dy - xmax;
	}else{
	  dy = dy + xmax;
	}
      if (!(dz < l2 && dz > -l2))
	if (dz > l2){
	  dz = dz - xmax;
	}else{
	  dz = dz + xmax;
	}

      dn2 = dx * dx + dy * dy + dz * dz;
      if (dn2 < cutoff_radius2){
	tmp2 = 2.e0 / dn2 / dn2 / dn2 / dn2 / dn2 / dn2 / dn2
	  - 1.e0 / dn2 / dn2 / dn2 / dn2;
	f = tmp2 * 24.e0;
	C[i*3]   += f * dx;
	C[i*3+1] += f * dy;
	C[i*3+2] += f * dz;
	C[j*3]   -= f * dx;
	C[j*3+1] -= f * dy;
	C[j*3+2] -= f * dz;
	
      }
      /*
      dn2 = (dx * dx + dy * dy + dz * dz) * 1;
      if (dn2 > exclude_radius2 && dn2 < cutoff_radius2){
	adn2 = 1.e0 / dn2;
	tmp2 = adn2 * adn2 * adn2;
	f = 24.e0 * adn2 * tmp2 * (2.e0 * tmp2 - 1.e0);
	fx += f * dx;
	fy += f * dy;
	fz += f * dz;
	
	}*/
    }
  }
}

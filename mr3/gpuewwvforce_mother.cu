#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#include <cutil.h>

#include <gpuewwvforce_kernel.cu>
////////////////////////////////////////////////////////////////////////////////
extern "C"
void gpuewwvforce_  (double*, int, double*, int, double*, double *);
void gpuewwvforce__ (double*, int, double*, int, double*, double *);

extern "C"
void computeGold_d(double*, const double*, const double*, int , int);
////////////////////////////////////////////////////////////////////////////////
//! 
////////////////////////////////////////////////////////////////////////////////
#if 0
extern "C"
void get_cputime(double *laptime, double *sprittime)
{
  struct timeval tv;
  struct timezone tz;
  double sec,microsec;

  gettimeofday(&tv, &tz);
  sec=tv.tv_sec;
  microsec=tv.tv_usec;

  *sprittime = sec + microsec * 1e-6 - *laptime;
  *laptime = sec + microsec * 1e-6;
}
#endif


extern "C"
void gpuewwvforce_ (double* xq, int num_atm, double* g_vec, int num_k, 
		    double* force, double *tpot)
{
#ifndef CUDA_SDK_2
   CUT_DEVICE_INIT();
#endif
//   CUT_CHECK_DEVICE();

//   double tpot[1];
   unsigned int size_A = ((num_atm+THD-1)/THD*THD) * 4;
   unsigned int mem_size_A = sizeof(float) * size_A;
   float* xq_float = (float*) malloc(mem_size_A);

   unsigned int size_B = ((num_k+THD-1)/THD*THD) * 4;
   unsigned int mem_size_B = sizeof(float) * size_B;
   float* gv_float = (float*) malloc(mem_size_B);

   unsigned int size_C = ((num_k+THD-1)/THD*THD) * 3;
   unsigned int mem_size_C = sizeof(float) * size_C;
   float* pot_float = (float*) malloc(mem_size_C);

   unsigned int size_D = ((num_atm+THD-1)/THD*THD) * 3;
   unsigned int mem_size_D = sizeof(float) * size_D;
   float* f_float = (float*) malloc(mem_size_D);
   //   unsigned int mem_size_E = sizeof(double) * size_D;
   //   double* force_double = (double*) malloc(mem_size_E);

   //double stime,ltime;

   for (int i = 0; i < size_A ; i++){
     if(i<num_atm*4) xq_float[i] = (float)xq[i];
     else            xq_float[i] = 0.0f;
   }
   
   for (int i = 0; i < size_B; i++){
     if(i<num_k*4) gv_float[i] = (float)g_vec[i];
     else          gv_float[i] = 0.0f;
     //printf("%16.6f %d \n",gv_float[i],i);
   }
   /*   for (int i = 0; i < num_k; i++)
     {
       printf("%16.6f %d \n",gv_float[i*4+3],i);
       }*/

   for (int i = 0; i < size_D; i++)
     {
       f_float[i] = 0.e0;
       //       force_double[i] = 0.e0;
     }

   //get_cputime(&ltime,&stime);

   float* d_A;
   CUDA_SAFE_CALL(hipMalloc((void**) &d_A, mem_size_A));
   CUDA_SAFE_CALL(hipMemcpy(d_A, xq_float, mem_size_A,hipMemcpyHostToDevice) );
   float* d_B;
   CUDA_SAFE_CALL(hipMalloc((void**) &d_B, mem_size_B));
   CUDA_SAFE_CALL(hipMemcpy(d_B, gv_float, mem_size_B,hipMemcpyHostToDevice) );
   float* d_C;
   CUDA_SAFE_CALL(hipMalloc((void**) &d_C, mem_size_C));
   float* d_D;
   CUDA_SAFE_CALL(hipMalloc((void**) &d_D, mem_size_D));

   dim3 threads(THD);
   dim3 grid((num_k+THD-1) / THD);
   gpuewwvforce_kernel<<< grid, threads >>>(d_C, d_A, d_B, num_atm);
   CUT_CHECK_ERROR("Kernel execution failed");
#if 1
   CUDA_SAFE_CALL(hipMemcpy(pot_float, d_C, mem_size_C,hipMemcpyDeviceToHost) );
   *tpot=0.0;
   for(int i=0;i<num_k;i++) *tpot+=pot_float[i*3+2];
   (*tpot)*=0.5;
   //   printf("tpot=%e 0:%f\n",*tpot);
#endif

   dim3 grid2((num_atm+THD-1) / THD);
   gpuewwvforce_kernel2<<< grid2, threads >>>(d_C, d_B, d_A, d_D, num_k);

   CUT_CHECK_ERROR("Kernel execution failed");
   CUDA_SAFE_CALL(hipMemcpy(f_float, d_D, mem_size_D,hipMemcpyDeviceToHost) );

   //get_cputime(&ltime,&stime);

   //printf("GPU  Processing time: %10.3f (sec)\n", stime);

   // host cumputation ////////////////////////////////////////

   //   get_cputime(&ltime,&stime);
   
   //   computeGold_d(force_double, g_vec, xq, num_atm, num_k);
   
   //   get_cputime(&ltime,&stime);
   
   //   printf("HOST Processing time: %10.3f (sec)\n", stime);
   // post preccess  ////////////////////////////////////////////
   
   //double err;
   for (int i = 0; i < num_atm; ++i){
     //printf("%16.6f %16.6f %d \n",f_float[i*3],force_double[i*3],i);
     
     force[i*3]   = (double)f_float[i*3];
     force[i*3+1] = (double)f_float[i*3+1];
     force[i*3+2] = (double)f_float[i*3+2];
     /*
     err += fabs((force_double[i*3] - (double)f_float[i*3]) 
		 / force_double[i*3]);
     err += fabs((force_double[i*3+1] - (double)f_float[i*3+1]) 
		 / force_double[i*3+1]);
     err += fabs((force_double[i*3+2] - (double)f_float[i*3+2]) 
     / force_double[i*3+2]);*/
     /*
     force[i*3]   = force_double[i*3];
     force[i*3+1] = force_double[i*3+1];
     force[i*3+2] = force_double[i*3+2];
     */
   }
   //err = err / (3.e0 * (double)num_atm) * 100.e0;
   //printf("err : %20.8f  \n",err);

   //printf("GPU : %20.8f  \n",sum_gpu);
   //printf("GPU : %20.8f \n",force[0]);
   //printf("HOST: %20.8f \n",force_double[0]);

   CUDA_SAFE_CALL(hipFree(d_A));
   CUDA_SAFE_CALL(hipFree(d_B));
   CUDA_SAFE_CALL(hipFree(d_C));
   CUDA_SAFE_CALL(hipFree(d_D));

   free(xq_float);
   free(gv_float);
   free(f_float);
   free(pot_float);
   //free(force_double);
}

extern "C"
void
gpuewwvforce__ (double* xq, int* num_atm, double* g_vec, int* num_k, 
		double* force, double *tpot)
{
  gpuewwvforce_ (xq,*num_atm,g_vec,*num_k,force,tpot);
}

extern "C"
void
computeGold_d(double* C, const double* A, const double* B, int num_atm, int num_k)
{
  //A : g_vec
  //B : xq

  double kr, qsin, qcos, tmp;

  unsigned int size_sc = num_atm;
  unsigned int mem_size_sc = sizeof(double) * size_sc;
  double* sin_theta = (double*) malloc(mem_size_sc);
  double* cos_theta = (double*) malloc(mem_size_sc);

  for (unsigned int i = 0; i < num_atm; ++i){
    C[i*3]   = 0.e0;
    C[i*3+1] = 0.e0;
    C[i*3+2] = 0.e0;
  }

  for (unsigned int i = 0; i < num_k; ++i){
    qsin = 0.e0;
    qcos = 0.e0;
    for (unsigned int j = 0; j < num_atm; ++j){
      kr = A[i*4]   * B[j*4]
	 + A[i*4+1] * B[j*4+1]
	 + A[i*4+2] * B[j*4+2];
      sin_theta[j] = sin(kr);
      cos_theta[j] = cos(kr);
      qsin += B[j*4+3] * sin_theta[j];
      qcos += B[j*4+3] * cos_theta[j];
    }
    //C[i*3]   = qsin;
    //C[i*3+1] = qcos;
    //C[i*3+2] = A[i*4+3] * ( qsin * qsin + qcos * qcos );
    qsin *= A[i*4+3];
    qcos *= A[i*4+3];
    for (unsigned int j = 0; j < num_atm; ++j){
      tmp = qcos * sin_theta[j] - qsin * cos_theta[j];
      C[j*3]   += tmp * A[i*4];
      C[j*3+1] += tmp * A[i*4+1];
      C[j*3+2] += tmp * A[i*4+2];
    }
  }

  for (unsigned int i = 0; i < num_atm; ++i){
    C[i*3]   *= B[i*4+3];
    C[i*3+1] *= B[i*4+3];
    C[i*3+2] *= B[i*4+3];
  }

  free(sin_theta);
  free(cos_theta);

}
